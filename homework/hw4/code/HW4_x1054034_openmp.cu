
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#define NUM_GPUS 2

const int INF = 10000000;
int *host_D;
int *dev_D[NUM_GPUS];
int n, m;

void Input(char *inFileName) {
	FILE *infile = fopen(inFileName, "r");
	setvbuf(infile, new char[1 << 20], _IOFBF, 1 << 20);
	fscanf(infile, "%d %d", &n, &m);
    host_D = (int*)malloc(n * n * sizeof(int));
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (i == j) host_D[i * n + j] = 0;
			else host_D[i * n + j] = INF;
		}
	}
	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		host_D[(a - 1) * n + (b - 1)] = v;
	}
	fclose(infile);
}

void Output(char *outFileName) {
	FILE *outfile = fopen(outFileName, "w");
	setvbuf(outfile, new char[1 << 20], _IOFBF, 1 << 20);
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (host_D[i * n + j] >= INF) fprintf(outfile, "INF ");
			else fprintf(outfile, "%d ", host_D[i * n + j]);
		}
		fprintf(outfile, "\n");
	}        
	fclose(outfile);
}

__global__ void func1(int n, int B, int k, int* arr) {
	extern __shared__ int shared_memory[];
    int* dBlock = shared_memory;
    int i = threadIdx.x / B;
    int j = threadIdx.x % B;
	int x = i + k * B;
	int y = j + k * B;
    dBlock[threadIdx.x] = (x < n && y < n)? arr[x * n + y]: INF;
    for (int l = 0; l < B; l++) {
        __syncthreads();
        int temp = dBlock[(i * B) + l] + dBlock[(l * B) + j];
        if (dBlock[threadIdx.x] > temp) {
            dBlock[threadIdx.x] = temp;
        }
    }
	if (x < n && y < n) arr[x * n + y] = dBlock[threadIdx.x];
}

__global__ void func2(int n, int B, int k, int* arr) {
	if (blockIdx.x == k) return;
	extern __shared__ int shared_memory[];
	int* dBlock = shared_memory;
    int* cBlock = &shared_memory[B * B];
    int i = threadIdx.x / B;
    int j = threadIdx.x % B;
	int x = i + k * B;
	int y = j + k * B;
    dBlock[threadIdx.x] = (x < n && y < n)? arr[x * n + y]: INF;
	if (blockIdx.y != 0) x = i + blockIdx.x * B;
	if (blockIdx.y == 0) y = j + blockIdx.x * B;
    cBlock[threadIdx.x] = (x < n && y < n)? arr[x * n + y]: INF;
    for (int l = 0; l < B; l++) {
        __syncthreads();
        int temp = (blockIdx.y == 0)? dBlock[i * B + l] + cBlock[l * B + j]: cBlock[i * B + l] + dBlock[l * B + j];
        if (cBlock[threadIdx.x] > temp) {
            cBlock[threadIdx.x] = temp;
        }
    }
    if (x < n && y < n) arr[x * n + y] = cBlock[threadIdx.x];
}

__global__ void func3(int n, int B, int k, int* arr, int offset) {
	if ((blockIdx.x + offset == k) || blockIdx.y == k) return;
	extern __shared__ int shared_memory[];
    int* dyBlock = shared_memory;
    int* dxBlock = &shared_memory[B * B];
    int i = threadIdx.x / B;
    int j = threadIdx.x % B;
	int x = i + k * B;
	int y = j + blockIdx.y * B;
    dxBlock[threadIdx.x] = (x < n && y < n)? arr[x * n + y]: INF;
	x = i + (blockIdx.x + offset) * B;
	y = j + k * B;
    dyBlock[threadIdx.x] = (x < n && y < n)? arr[x * n + y]: INF;
	x = i + (blockIdx.x + offset) * B;
	y = j + blockIdx.y * B;
    int dist = (x < n && y < n)? arr[x * n + y]: INF;
    __syncthreads();
    for (int l = 0; l < B; l++) {
        int temp = dyBlock[i * B + l] + dxBlock[l * B + j];
        if (dist > temp) {
            dist = temp;
        }
    }
	if (x < n && y < n) arr[x * n + y] = dist;
}

void Block(int B) {
	omp_set_num_threads(NUM_GPUS);
	int round = (n + B - 1) / B;
	int offset = (round + NUM_GPUS - 1) / NUM_GPUS;
	int num[2] = { offset * B * n, n * n - offset * B * n };
	dim3 bk1(1, 1);
    dim3 bk2(round, 2);
    dim3 bk3(offset, round);
    int gputhreads = B * B;
	#pragma omp parallel 
	{
		unsigned int thread_id = omp_get_thread_num();
		hipSetDevice(thread_id);
		hipDeviceEnablePeerAccess(!thread_id, 0);
		hipMalloc(&dev_D[thread_id], n * n * sizeof(int));
		hipMemcpy(dev_D[thread_id], host_D, n * n * sizeof(int), hipMemcpyHostToDevice);
		for (int k = 0; k < round; k++) {
			func1<<<bk1, gputhreads, gputhreads * sizeof(int)>>>(n, B, k, dev_D[thread_id]);
			func2<<<bk2, gputhreads, 2 * gputhreads * sizeof(int)>>>(n, B, k, dev_D[thread_id]);
			func3<<<bk3, gputhreads, 2 * gputhreads * sizeof(int)>>>(n, B, k, dev_D[thread_id], thread_id * offset);
			#pragma omp barrier
			if (thread_id == 0) hipMemcpyPeer(dev_D[1], 1, dev_D[0], 0, num[0] * sizeof(int));
			else hipMemcpyPeer(dev_D[0] + num[0], 0, dev_D[1] + num[0], 1, num[1] * sizeof(int));
			#pragma omp barrier
		}
	}
    hipMemcpy(host_D, dev_D[0], n * n * sizeof(int), hipMemcpyDeviceToHost);
}

int main(int argc, char **argv) {
	Input(argv[1]);
	int B = atoi(argv[3]);
	Block(B);
	Output(argv[2]);
    return 0;
}
